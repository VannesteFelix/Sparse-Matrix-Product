#include "hip/hip_runtime.h"
//#include <hipsparse.h>
//#include <stdio.h>
//#include <time.h>
//#include <sys/time.h>

//// error check macros
//#define CUSPARSE_CHECK(x) {hipsparseStatus_t _c=x; if (_c != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse fail: %d, line: %d\n", (int)_c, __LINE__); exit(-1);}}

//#define cudaCheckErrors(msg) \
//do { \
//    hipError_t __err = hipGetLastError(); \
//    if (__err != hipSuccess) { \
//        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
//            msg, hipGetErrorString(__err), \
//            __FILE__, __LINE__); \
//        fprintf(stderr, "*** FAILED - ABORTING\n"); \
//        exit(1); \
//    } \
//} while (0)


//double timerval()
//{
//    struct timeval st;
//    gettimeofday(&st, NULL);
//    return (st.tv_sec+st.tv_usec*1e-6);
//}

//// perform sparse-matrix multiplication C=AxB
//int main(){
//double avg_time = 0, s_time, e_time;

//hipsparseStatus_t stat;
//hipsparseHandle_t hndl;
//hipsparseMatDescr_t descrA, descrB, descrC;
//int *csrRowPtrA, *csrRowPtrB, *csrRowPtrC, *csrColIndA, *csrColIndB, *csrColIndC;
//int *h_csrRowPtrA, *h_csrRowPtrB, *h_csrRowPtrC, *h_csrColIndA, *h_csrColIndB, *h_csrColIndC,*pos;
//float *csrValA, *csrValB, *csrValC, *h_csrValA, *h_csrValB, *h_csrValC;
//int nnzA, nnzB, nnzC;
//int m=4,n,k,loop;
//int i,j;
//int iterations;
//for (iterations=0;iterations<10;iterations++)
//{
//    m *=2;
//    n = m;
//    k = m;
//    //density of the sparse matrix to be created. Assume 5% density.
//    double dense_const = 0.05;
//    int temp5, temp6,temp3,temp4;
//    int density=(m*n)*(dense_const);
//    nnzA = density;
//    nnzB = density;
//    h_csrRowPtrA = (int *)malloc((m+1)*sizeof(int));
//    h_csrRowPtrB = (int *)malloc((n+1)*sizeof(int));
//    h_csrColIndA = (int *)malloc(density*sizeof(int));
//    h_csrColIndB = (int *)malloc(density*sizeof(int));
//    h_csrValA  = (float *)malloc(density*sizeof(float));
//    h_csrValB  = (float *)malloc(density*sizeof(float));
//    if ((h_csrRowPtrA == NULL) || (h_csrRowPtrB == NULL) || (h_csrColIndA == NULL) || (h_csrColIndB == NULL) || (h_csrValA == NULL) || (h_csrValB == NULL))
//    {printf("malloc fail\n"); return -1;}

//    //position array for random initialisation of positions in input matrix
//    pos= (int *)calloc((m*n), sizeof(int));
//    int temp,temp1;

//    //  printf("the density is %d\n",density);
//    //  printf("check 1:\n");

//    //randomly initialise positions
//    for(i=0;i<density;i++)
//    {
//        temp1=rand()%(m*n);
//        pos[i]=temp1;

//    }
//    //  printf("check 2:\n");

//    //sort the 'pos' array
//    for (i = 0 ; i < density; i++) {
//        int d = i;
//        int t;

//        while ( d > 0 && pos[d] < pos[d-1]) {
//            t          = pos[d];
//            pos[d]   = pos[d-1];
//            pos[d-1] = t;
//            d--;
//        }
//    }
//    // initialise with non zero elements and extract column and row ptr vector
//    j=1;
//    //ja[0]=1;

//    int p=0;
//    int f=0;

//    for(i = 0; i < density; i++)
//    {
//        temp=pos[i];
//         h_csrValA[f] = rand();
//         h_csrValB[f] = rand();
//         h_csrColIndA[f] = temp%m;
//         h_csrColIndB[f] = temp%m;
//        f++;
//        p++;
//        temp5= pos[i];
//        temp6=pos[i+1];
//        temp3=temp5-(temp5%m);
//        temp4=temp6-(temp6%m);

//        if(!(temp3== temp4))
//        {
//            if((temp3+m==temp6))
//            {}
//            else
//            {
//                h_csrRowPtrA[j]=p;
//                h_csrRowPtrB[j]=p;
//                j++;
//            }

//        }
//    }

//    // transfer data to device

//    hipMalloc(&csrRowPtrA, (m+1)*sizeof(int));
//    hipMalloc(&csrRowPtrB, (n+1)*sizeof(int));
//    hipMalloc(&csrColIndA, density*sizeof(int));
//    hipMalloc(&csrColIndB, density*sizeof(int));
//    hipMalloc(&csrValA, density*sizeof(float));
//    hipMalloc(&csrValB, density*sizeof(float));
//    cudaCheckErrors("hipMalloc fail");
//    hipMemcpy(csrRowPtrA, h_csrRowPtrA, (m+1)*sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(csrRowPtrB, h_csrRowPtrB, (n+1)*sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(csrColIndA, h_csrColIndA, density*sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(csrColIndB, h_csrColIndB, density*sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(csrValA, h_csrValA, density*sizeof(float), hipMemcpyHostToDevice);
//    hipMemcpy(csrValB, h_csrValB, density*sizeof(float), hipMemcpyHostToDevice);
//    cudaCheckErrors("hipMemcpy fail");

//    // set cusparse matrix types
//    CUSPARSE_CHECK(hipsparseCreate(&hndl));
//    stat = hipsparseCreateMatDescr(&descrA);
//    CUSPARSE_CHECK(stat);
//    stat = hipsparseCreateMatDescr(&descrB);
//    CUSPARSE_CHECK(stat);
//    stat = hipsparseCreateMatDescr(&descrC);
//    CUSPARSE_CHECK(stat);
//    stat = hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
//    CUSPARSE_CHECK(stat);
//    stat = hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
//    CUSPARSE_CHECK(stat);
//    stat = hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
//    CUSPARSE_CHECK(stat);
//    stat = hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
//    CUSPARSE_CHECK(stat);
//    stat = hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
//    CUSPARSE_CHECK(stat);
//    stat = hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
//    CUSPARSE_CHECK(stat);
//    hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
//    hipsparseOperation_t transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

//    // figure out size of C
//    int baseC;
//    // nnzTotalDevHostPtr points to host memory
//    int *nnzTotalDevHostPtr = &nnzC;
//    stat = hipsparseSetPointerMode(hndl, HIPSPARSE_POINTER_MODE_HOST);
//    CUSPARSE_CHECK(stat);
//    hipMalloc((void**)&csrRowPtrC, sizeof(int)*(m+1));
//    cudaCheckErrors("hipMalloc fail");

//    s_time=timerval();

//    stat = hipsparseXcsrgemmNnz(hndl, transA, transB, m, n, k,
//    descrA, nnzA, csrRowPtrA, csrColIndA,
//    descrB, nnzB, csrRowPtrB, csrColIndB,
//    descrC, csrRowPtrC, nnzTotalDevHostPtr );
//    CUSPARSE_CHECK(stat);
//    if (NULL != nnzTotalDevHostPtr){
//    nnzC = *nnzTotalDevHostPtr;}
//    else{
//    hipMemcpy(&nnzC, csrRowPtrC+m, sizeof(int), hipMemcpyDeviceToHost);
//    hipMemcpy(&baseC, csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
//    cudaCheckErrors("hipMemcpy fail");
//    nnzC -= baseC;}
//    hipMalloc((void**)&csrColIndC, sizeof(int)*nnzC);
//    hipMalloc((void**)&csrValC, sizeof(float)*nnzC);
//    cudaCheckErrors("hipMalloc fail");
//    // perform multiplication C = A*B

//    for(loop=0;loop<1000;loop++)
//    {
//        stat = hipsparseScsrgemm(hndl, transA, transB, m, n, k,
//        descrA, nnzA,
//        csrValA, csrRowPtrA, csrColIndA,
//        descrB, nnzB,
//        csrValB, csrRowPtrB, csrColIndB,
//        descrC,
//        csrValC, csrRowPtrC, csrColIndC);
//        CUSPARSE_CHECK(stat);
//    }

//    e_time=timerval();

//    avg_time=avg_time/1000;
//    // copy result (C) back to host
//    h_csrRowPtrC = (int *)malloc((m+1)*sizeof(int));
//    h_csrColIndC = (int *)malloc(nnzC *sizeof(int));
//    h_csrValC  = (float *)malloc(nnzC *sizeof(float));
//    if ((h_csrRowPtrC == NULL) || (h_csrColIndC == NULL) || (h_csrValC == NULL))
//    {printf("malloc fail\n"); return -1;}
//    hipMemcpy(h_csrRowPtrC, csrRowPtrC, (m+1)*sizeof(int), hipMemcpyDeviceToHost);
//    hipMemcpy(h_csrColIndC, csrColIndC,  nnzC*sizeof(int), hipMemcpyDeviceToHost);
//    hipMemcpy(h_csrValC, csrValC, nnzC*sizeof(float), hipMemcpyDeviceToHost);
//    cudaCheckErrors("hipMemcpy fail");

//    printf ("\n Input size: %d x %d ,Time: %lf and density is %d \n", m,n, avg_time, density);

//    hipFree(csrRowPtrC);
//    hipFree(csrColIndC);
//    hipFree(csrValC);

//    hipFree(csrRowPtrA);
//    hipFree(csrColIndA);
//    hipFree(csrValA);

//    hipFree(csrRowPtrB);
//    hipFree(csrColIndB);
//    hipFree(csrValB);

//    free(h_csrRowPtrC);
//    free(h_csrColIndC);
//    free(h_csrValC);

//    free(h_csrRowPtrA);
//    free(h_csrColIndA);
//    free(h_csrValA);

//    free(h_csrRowPtrB);
//    free(h_csrColIndB);
//    free(h_csrValB);
//}
//return 0;
//}








////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////








//#include <stdio.h>
//#include <stdlib.h>
//#include <iostream>
//#include <assert.h>

//#include "Utilities.cuh"

//#include <hip/hip_runtime.h>
//#include <hipsparse.h>

///********/
///* MAIN */
///********/
//int main()
//{
//    // --- Initialize cuSPARSE
//    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

//    /**************************/
//    /* SETTING UP THE PROBLEM */
//    /**************************/
//    const int N     = 4;                // --- Number of rows and columns

//    // --- Host side dense matrices
//    double *h_A_dense = (double*)malloc(N * N * sizeof(*h_A_dense));
//    double *h_B_dense = (double*)malloc(N * N * sizeof(*h_B_dense));
//    double *h_C_dense = (double*)malloc(N * N * sizeof(*h_C_dense));

//    // --- Column-major ordering
//    h_A_dense[0] = 0.4612;  h_A_dense[4] = -0.0006;     h_A_dense[8]  = 0.3566;     h_A_dense[12] = 0.0;
//    h_A_dense[1] = -0.0006; h_A_dense[5] = 0.4640;      h_A_dense[9]  = 0.0723;     h_A_dense[13] = 0.0;
//    h_A_dense[2] = 0.3566;  h_A_dense[6] = 0.0723;      h_A_dense[10] = 0.7543;     h_A_dense[14] = 0.0;
//    h_A_dense[3] = 0.;      h_A_dense[7] = 0.0;         h_A_dense[11] = 0.0;        h_A_dense[15] = 0.1;

//    // --- Column-major ordering
//    h_B_dense[0] = 0.;      h_B_dense[4] = 0.;          h_B_dense[8]  = 1.;         h_B_dense[12] = 0.;
//    h_B_dense[1] = 1.;      h_B_dense[5] = 0.;          h_B_dense[9]  = 0.;         h_B_dense[13] = 0.;
//    h_B_dense[2] = 0.;      h_B_dense[6] = 1.;          h_B_dense[10] = 0.;         h_B_dense[14] = 0.;
//    h_B_dense[3] = 0.;      h_B_dense[7] = 0.;          h_B_dense[11] = 0.;         h_B_dense[15] = 1.;

//    // --- Create device arrays and copy host arrays to them
//    double *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, N * N * sizeof(*d_A_dense)));
//    double *d_B_dense;  gpuErrchk(hipMalloc(&d_B_dense, N * N * sizeof(*d_B_dense)));
//    double *d_C_dense;  gpuErrchk(hipMalloc(&d_C_dense, N * N * sizeof(*d_C_dense)));
//    gpuErrchk(hipMemcpy(d_A_dense, h_A_dense, N * N * sizeof(*d_A_dense), hipMemcpyHostToDevice));
//    gpuErrchk(hipMemcpy(d_B_dense, h_B_dense, N * N * sizeof(*d_B_dense), hipMemcpyHostToDevice));

//    // --- Descriptor for sparse matrix A
//    hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
//    cusparseSafeCall(hipsparseSetMatType     (descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
//    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));

//    // --- Descriptor for sparse matrix B
//    hipsparseMatDescr_t descrB;      cusparseSafeCall(hipsparseCreateMatDescr(&descrB));
//    cusparseSafeCall(hipsparseSetMatType     (descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
//    cusparseSafeCall(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ONE));

//    // --- Descriptor for sparse matrix C
//    hipsparseMatDescr_t descrC;      cusparseSafeCall(hipsparseCreateMatDescr(&descrC));
//    cusparseSafeCall(hipsparseSetMatType     (descrC, HIPSPARSE_MATRIX_TYPE_GENERAL));
//    cusparseSafeCall(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ONE));

//    int nnzA = 0;                           // --- Number of nonzero elements in dense matrix A
//    int nnzB = 0;                           // --- Number of nonzero elements in dense matrix B

//    const int lda = N;                      // --- Leading dimension of dense matrix

//    // --- Device side number of nonzero elements per row of matrix A
//    int *d_nnzPerVectorA;   gpuErrchk(hipMalloc(&d_nnzPerVectorA, N * sizeof(*d_nnzPerVectorA)));
//    cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, N, N, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA));

//    // --- Device side number of nonzero elements per row of matrix B
//    int *d_nnzPerVectorB;   gpuErrchk(hipMalloc(&d_nnzPerVectorB, N * sizeof(*d_nnzPerVectorB)));
//    cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, N, N, descrB, d_B_dense, lda, d_nnzPerVectorB, &nnzB));

//    // --- Host side number of nonzero elements per row of matrix A
//    int *h_nnzPerVectorA = (int *)malloc(N * sizeof(*h_nnzPerVectorA));
//    gpuErrchk(hipMemcpy(h_nnzPerVectorA, d_nnzPerVectorA, N * sizeof(*h_nnzPerVectorA), hipMemcpyDeviceToHost));

//    // --- Host side number of nonzero elements per row of matrix B
//    int *h_nnzPerVectorB = (int *)malloc(N * sizeof(*h_nnzPerVectorB));
//    gpuErrchk(hipMemcpy(h_nnzPerVectorB, d_nnzPerVectorB, N * sizeof(*h_nnzPerVectorB), hipMemcpyDeviceToHost));

//    printf("Number of nonzero elements in dense matrix A = %i\n\n", nnzA);
//    for (int i = 0; i < N; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, h_nnzPerVectorA[i]);
//    printf("\n");

//    printf("Number of nonzero elements in dense matrix B = %i\n\n", nnzB);
//    for (int i = 0; i < N; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, h_nnzPerVectorB[i]);
//    printf("\n");

//    // --- Device side sparse matrix
//    double *d_A;            gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
//    double *d_B;            gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));

//    int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (N + 1) * sizeof(*d_A_RowIndices)));
//    int *d_B_RowIndices;    gpuErrchk(hipMalloc(&d_B_RowIndices, (N + 1) * sizeof(*d_B_RowIndices)));
//    int *d_C_RowIndices;    gpuErrchk(hipMalloc(&d_C_RowIndices, (N + 1) * sizeof(*d_C_RowIndices)));
//    int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
//    int *d_B_ColIndices;    gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));

//    cusparseSafeCall(hipsparseDdense2csr(handle, N, N, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices));
//    cusparseSafeCall(hipsparseDdense2csr(handle, N, N, descrB, d_B_dense, lda, d_nnzPerVectorB, d_B, d_B_RowIndices, d_B_ColIndices));

//    // --- Host side sparse matrices
//    double *h_A = (double *)malloc(nnzA * sizeof(*h_A));
//    double *h_B = (double *)malloc(nnzB * sizeof(*h_B));
//    int *h_A_RowIndices = (int *)malloc((N + 1) * sizeof(*h_A_RowIndices));
//    int *h_A_ColIndices = (int *)malloc(nnzA * sizeof(*h_A_ColIndices));
//    int *h_B_RowIndices = (int *)malloc((N + 1) * sizeof(*h_B_RowIndices));
//    int *h_B_ColIndices = (int *)malloc(nnzB * sizeof(*h_B_ColIndices));
//    int *h_C_RowIndices = (int *)malloc((N + 1) * sizeof(*h_C_RowIndices));
//    gpuErrchk(hipMemcpy(h_A, d_A, nnzA * sizeof(*h_A), hipMemcpyDeviceToHost));
//    gpuErrchk(hipMemcpy(h_A_RowIndices, d_A_RowIndices, (N + 1) * sizeof(*h_A_RowIndices), hipMemcpyDeviceToHost));
//    gpuErrchk(hipMemcpy(h_A_ColIndices, d_A_ColIndices, nnzA * sizeof(*h_A_ColIndices), hipMemcpyDeviceToHost));
//    gpuErrchk(hipMemcpy(h_B, d_B, nnzB * sizeof(*h_B), hipMemcpyDeviceToHost));
//    gpuErrchk(hipMemcpy(h_B_RowIndices, d_B_RowIndices, (N + 1) * sizeof(*h_B_RowIndices), hipMemcpyDeviceToHost));
//    gpuErrchk(hipMemcpy(h_B_ColIndices, d_B_ColIndices, nnzB * sizeof(*h_B_ColIndices), hipMemcpyDeviceToHost));

//    printf("\nOriginal matrix A in CSR format\n\n");
//    for (int i = 0; i < nnzA; ++i) printf("A[%i] = %f ", i, h_A[i]); printf("\n");

//    printf("\nOriginal matrix B in CSR format\n\n");
//    for (int i = 0; i < nnzB; ++i) printf("B[%i] = %f ", i, h_B[i]); printf("\n");

//    printf("\n");
//    for (int i = 0; i < (N + 1); ++i) printf("h_A_RowIndices[%i] = %i \n", i, h_A_RowIndices[i]); printf("\n");

//    printf("\n");
//    for (int i = 0; i < (N + 1); ++i) printf("h_B_RowIndices[%i] = %i \n", i, h_B_RowIndices[i]); printf("\n");

//    printf("\n");
//    for (int i = 0; i < nnzA; ++i) printf("h_A_ColIndices[%i] = %i \n", i, h_A_ColIndices[i]);

//    printf("\n");
//    for (int i = 0; i < nnzB; ++i) printf("h_B_ColIndices[%i] = %i \n", i, h_B_ColIndices[i]);

//    // --- Performing the matrix - matrix multiplication
//    int baseC, nnzC = 0;
//    // nnzTotalDevHostPtr points to host memory
//    int *nnzTotalDevHostPtr = &nnzC;

//    cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

//    cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, N, descrB, nnzB,
//                                         d_B_RowIndices, d_B_ColIndices, descrA, nnzA, d_A_RowIndices, d_A_ColIndices, descrC, d_C_RowIndices,
//                                         nnzTotalDevHostPtr));
//    if (NULL != nnzTotalDevHostPtr) nnzC = *nnzTotalDevHostPtr;
//    else {
//        gpuErrchk(hipMemcpy(&nnzC,  d_C_RowIndices + N, sizeof(int), hipMemcpyDeviceToHost));
//        gpuErrchk(hipMemcpy(&baseC, d_C_RowIndices,     sizeof(int), hipMemcpyDeviceToHost));
//        nnzC -= baseC;
//    }
//    int *d_C_ColIndices;    gpuErrchk(hipMalloc(&d_C_ColIndices, nnzC * sizeof(int)));
//    double *d_C;            gpuErrchk(hipMalloc(&d_C, nnzC * sizeof(double)));
//    double *h_C = (double *)malloc(nnzC * sizeof(*h_C));
//    int *h_C_ColIndices = (int *)malloc(nnzC * sizeof(*h_C_ColIndices));
//    cusparseSafeCall(hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, N, descrB, nnzB,
//                                      d_B, d_B_RowIndices, d_B_ColIndices, descrA, nnzA, d_A, d_A_RowIndices, d_A_ColIndices, descrC,
//                                      d_C, d_C_RowIndices, d_C_ColIndices));

//    cusparseSafeCall(hipsparseDcsr2dense(handle, N, N, descrC, d_C, d_C_RowIndices, d_C_ColIndices, d_C_dense, N));

//    gpuErrchk(hipMemcpy(h_C ,           d_C,            nnzC * sizeof(*h_C), hipMemcpyDeviceToHost));
//    gpuErrchk(hipMemcpy(h_C_RowIndices, d_C_RowIndices, (N + 1) * sizeof(*h_C_RowIndices), hipMemcpyDeviceToHost));
//    gpuErrchk(hipMemcpy(h_C_ColIndices, d_C_ColIndices, nnzC * sizeof(*h_C_ColIndices), hipMemcpyDeviceToHost));

//    printf("\nResult matrix C in CSR format\n\n");
//    for (int i = 0; i < nnzC; ++i) printf("C[%i] = %f ", i, h_C[i]); printf("\n");

//    printf("\n");
//    for (int i = 0; i < (N + 1); ++i) printf("h_C_RowIndices[%i] = %i \n", i, h_C_RowIndices[i]); printf("\n");

//    printf("\n");
//    for (int i = 0; i < nnzC; ++i) printf("h_C_ColIndices[%i] = %i \n", i, h_C_ColIndices[i]);

//    gpuErrchk(hipMemcpy(h_C_dense, d_C_dense, N * N * sizeof(double), hipMemcpyDeviceToHost));

//    for (int j = 0; j < N; j++) {
//        for (int i = 0; i < N; i++)
//            printf("%f \t", h_C_dense[i * N + j]);
//        printf("\n");
//        }
//}








////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////

#include <hipsparse.h>
#include <stdio.h>

#include <ctime>


//erfredrg

#define N 50000

// matrix generation and validation depends on these relationships:
#define SCL 2
#define K N
#define M (SCL*N)
// A: MxK  B: KxN  C: MxN

// error check macros
#define CUSPARSE_CHECK(x) {hipsparseStatus_t _c=x; if (_c != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse fail: %d, line: %d\n", (int)_c, __LINE__); exit(-1);}}

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

// perform sparse-matrix multiplication C=AxB
int main(){

    std::clock_t start;
    double duration;

    hipsparseStatus_t stat;
    hipsparseHandle_t hndl;
    hipsparseMatDescr_t descrA, descrB, descrC;
    int *csrRowPtrA, *csrRowPtrB, *csrRowPtrC, *csrColIndA, *csrColIndB, *csrColIndC;
    int *h_csrRowPtrA, *h_csrRowPtrB, *h_csrRowPtrC, *h_csrColIndA, *h_csrColIndB, *h_csrColIndC;
    float *csrValA, *csrValB, *csrValC, *h_csrValA, *h_csrValB, *h_csrValC;
    int nnzA, nnzB, nnzC;
    int m,n,k;
    m = M;
    n = N;
    k = K;


    ///////////////////////////     STEP 1      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                           generate A, B=2I                                 */
    /*
                                 A:
                                |1.0 0.0 0.0 ...|
                                |1.0 0.0 0.0 ...|
                                |0.0 1.0 0.0 ...|
                                |0.0 1.0 0.0 ...|
                                |0.0 0.0 1.0 ...|
                                |0.0 0.0 1.0 ...|
                                ...

                                B:
                                |2.0 0.0 0.0 ...|
                                |0.0 2.0 0.0 ...|
                                |0.0 0.0 2.0 ...|
                                ...                                               */
    /* -------------------------------------------------------------------------- */
    start = std::clock();

        nnzA = m;
        nnzB = n;
        h_csrRowPtrA = (int *)malloc((m+1)*sizeof(int));
        h_csrRowPtrB = (int *)malloc((n+1)*sizeof(int));
        h_csrColIndA = (int *)malloc(m*sizeof(int));
        h_csrColIndB = (int *)malloc(n*sizeof(int));
        h_csrValA  = (float *)malloc(m*sizeof(float));
        h_csrValB  = (float *)malloc(n*sizeof(float));

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("Host Malloc :                                   %f ms\n", duration);
    //------------------------------------------------------------------------------
    start = std::clock();

        if ((h_csrRowPtrA == NULL) || (h_csrRowPtrB == NULL) || (h_csrColIndA == NULL) || (h_csrColIndB == NULL) || (h_csrValA == NULL) || (h_csrValB == NULL))
        {printf("malloc fail\n"); return -1;}
        for (int i = 0; i < m; i++){
        h_csrValA[i] = 1.0f;
        h_csrRowPtrA[i] = i;
        h_csrColIndA[i] = i/SCL;
        if (i < n){
          h_csrValB[i] = 2.0f;
          h_csrRowPtrB[i] = i;
          h_csrColIndB[i] = i;}
        }
        h_csrRowPtrA[m] = m;
        h_csrRowPtrB[n] = n;


    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("CSR Matrix Generation :                         %f ms\n", duration);
    ///////////////////////////     STEP 2      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                      Allocate memory on the device                         */
    /*              and return a ptr of its memory emplacement                    */
    /* -------------------------------------------------------------------------- */
    start = std::clock();

        hipMalloc(&csrRowPtrA, (m+1)*sizeof(int));

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("hipMalloc csrRowPtrA :                         %f ms\n", duration);
    //------------------------------------------------------------------------------
    start = std::clock();

        hipMalloc(&csrRowPtrB, (n+1)*sizeof(int));
        hipMalloc(&csrColIndA, m*sizeof(int));
        hipMalloc(&csrColIndB, n*sizeof(int));
        hipMalloc(&csrValA, m*sizeof(float));
        hipMalloc(&csrValB, n*sizeof(float));


    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("hipMalloc csrRowPtrB|csrColIndA/B|csrValA/B :  %f ms\n", duration);
    ///////////////////////////     STEP 3      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                  Copy the data from the Host (CPU)                         */
    /*                      to the device (GPU)                                   */
    /* -------------------------------------------------------------------------- */
    start = std::clock();

        cudaCheckErrors("hipMalloc fail");
        hipMemcpy(csrRowPtrA, h_csrRowPtrA, (m+1)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrRowPtrB, h_csrRowPtrB, (n+1)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrColIndA, h_csrColIndA, m*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrColIndB, h_csrColIndB, n*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrValA, h_csrValA, m*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(csrValB, h_csrValB, n*sizeof(float), hipMemcpyHostToDevice);
        cudaCheckErrors("hipMemcpy fail");


    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("Copy Data from Host to Device :                 %f ms\n", duration);
    ///////////////////////////     STEP 4      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                      set cusparse matrix types                             */
    /*                             ?????                                          */
    /* -------------------------------------------------------------------------- */
    start = std::clock();

        CUSPARSE_CHECK(hipsparseCreate(&hndl));

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("hipsparseCreate(&hndl) :                         %f ms\n", duration);
    //------------------------------------------------------------------------------
    start = std::clock();

        stat = hipsparseCreateMatDescr(&descrA);
        CUSPARSE_CHECK(stat);
        stat = hipsparseCreateMatDescr(&descrB);
        CUSPARSE_CHECK(stat);
        stat = hipsparseCreateMatDescr(&descrC);
        CUSPARSE_CHECK(stat);

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("hipsparseCreateMatDescr(&descrA/B/C) :           %f ms\n", duration);
    //------------------------------------------------------------------------------
    start = std::clock();

        stat = hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
        CUSPARSE_CHECK(stat);
        stat = hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
        CUSPARSE_CHECK(stat);
        stat = hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
        CUSPARSE_CHECK(stat);
        stat = hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
        CUSPARSE_CHECK(stat);
        stat = hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
        CUSPARSE_CHECK(stat);
        stat = hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
        CUSPARSE_CHECK(stat);
        hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;


    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("Set cusparse matrix types :                     %f ms\n", duration);
    ///////////////////////////     STEP 5      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                             ??????                                         */
    /*                                                                            */
    /* -------------------------------------------------------------------------- */
    start = std::clock();

        // figure out size of C
        int baseC;

        // nnzTotalDevHostPtr points to host memory
        int *nnzTotalDevHostPtr = &nnzC;
        stat = hipsparseSetPointerMode(hndl, HIPSPARSE_POINTER_MODE_HOST);
        CUSPARSE_CHECK(stat);
        hipMalloc((void**)&csrRowPtrC, sizeof(int)*(m+1));
        cudaCheckErrors("hipMalloc fail");

    //------------------------------------------------------------------------------

        // ????
        stat = hipsparseXcsrgemmNnz(hndl, transA, transB, m, n, k,
            descrA, nnzA, csrRowPtrA, csrColIndA,
            descrB, nnzB, csrRowPtrB, csrColIndB,
            descrC, csrRowPtrC, nnzTotalDevHostPtr );
        CUSPARSE_CHECK(stat);

    //------------------------------------------------------------------------------

        //  ????
        if (NULL != nnzTotalDevHostPtr){
        nnzC = *nnzTotalDevHostPtr;}
        else{
        hipMemcpy(&nnzC, csrRowPtrC+m, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy fail");
        nnzC -= baseC;}
        hipMalloc((void**)&csrColIndC, sizeof(int)*nnzC);
        hipMalloc((void**)&csrValC, sizeof(float)*nnzC);
        cudaCheckErrors("hipMalloc fail");


    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("nnzTotalDevHostPtr points to host memory :      %f ms\n", duration);
    ///////////////////////////     STEP 6      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                Perform multiplication C = A*B                              */
    /*                                                                            */
    /* -------------------------------------------------------------------------- */
    start = std::clock();

        stat = hipsparseScsrgemm(hndl, transA, transB, m, n, k,
            descrA, nnzA,
            csrValA, csrRowPtrA, csrColIndA,
            descrB, nnzB,
            csrValB, csrRowPtrB, csrColIndB,
            descrC,
            csrValC, csrRowPtrC, csrColIndC);
        CUSPARSE_CHECK(stat);


    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("GPU calculation time :                          %f ms\n", duration);
    ///////////////////////////     STEP 7      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                  Copy result (C) back to host                              */
    /*                       & test & validate it                                 */
    /* -------------------------------------------------------------------------- */
    start = std::clock();

        // copy result (C) back to host
        h_csrRowPtrC = (int *)malloc((m+1)*sizeof(int));
        h_csrColIndC = (int *)malloc(nnzC *sizeof(int));
        h_csrValC  = (float *)malloc(nnzC *sizeof(float));
        if ((h_csrRowPtrC == NULL) || (h_csrColIndC == NULL) || (h_csrValC == NULL))
        {printf("malloc fail\n"); return -1;}
        hipMemcpy(h_csrRowPtrC, csrRowPtrC, (m+1)*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_csrColIndC, csrColIndC,  nnzC*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_csrValC, csrValC, nnzC*sizeof(float), hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy fail");

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("Copy GPU to CPU :                               %f ms\n", duration);
    //------------------------------------------------------------------------------

        // check result, C = 2A
        if (nnzC != m) {printf("invalid matrix size C: %d, should be: %d\n", nnzC, m); return -1;}
        for (int i = 0; i < m; i++){
        if (h_csrRowPtrA[i] != h_csrRowPtrC[i]) {printf("A/C row ptr mismatch at %d, A: %d, C: %d\n", i, h_csrRowPtrA[i], h_csrRowPtrC[i]); return -1;}
        if (h_csrColIndA[i] != h_csrColIndC[i]) {printf("A/C col ind mismatch at %d, A: %d, C: %d\n", i, h_csrColIndA[i], h_csrColIndC[i]); return -1;}
        if ((h_csrValA[i]*2.0f) != h_csrValC[i]) {printf("A/C value mismatch at %d, A: %f, C: %f\n", i, h_csrValA[i]*2.0f, h_csrValC[i]); return -1;}
        }
        printf("Success!\n");

    return 0;
}
