#include "hip/hip_runtime.h"
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>

#ifdef WINDOWS
#include <direct.h>
#define GetCurrentDir _getcwd
#else
#include <unistd.h>
#define GetCurrentDir getcwd
#endif


extern "C" {
#include "component/libraries/mmio.h"
}

///////////////////////////////////////////

#include <boost/numeric/ublas/triangular.hpp>
#include <boost/numeric/ublas/vector.hpp>
#include <boost/numeric/ublas/vector_proxy.hpp>
#include <boost/numeric/ublas/matrix_sparse.hpp>
#include <boost/numeric/ublas/operation_sparse.hpp>
#include <boost/numeric/ublas/lu.hpp>

#include "viennacl/scalar.hpp"
#include "viennacl/vector.hpp"
#include "viennacl/tools/timer.hpp"
#include "viennacl/coordinate_matrix.hpp"
#include "viennacl/compressed_matrix.hpp"
#include "viennacl/ell_matrix.hpp"
#include "viennacl/hyb_matrix.hpp"
#include "viennacl/sliced_ell_matrix.hpp"
#include "viennacl/linalg/prod.hpp"
#include "viennacl/linalg/norm_2.hpp"
#include "viennacl/io/matrix_market.hpp"

///////////////////////////////////////////

struct MyCSRMat
{
    int * I;        // ROW INDICES OF NZ
    int * J;        // COLUMN INDICES OF NZ
    double * val;   // VALUES OF NZ
    int nz;         // NON-ZERO
    int M;          // ROW
    int N;          // COLUMN
}myMat1,myMat2,myMat3;

// error check macros
#define CUSPARSE_CHECK(x) {hipsparseStatus_t _c=x; if (_c != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse fail: %d, line: %d\n", (int)_c, __LINE__); exit(-1);}}

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

char* appendCharToCharArray(char* array, char a)
{
    size_t len = strlen(array);

    char* ret = new char[len+2];

    strcpy(ret, array);
    ret[len] = a;
    ret[len+1] = '\0';

    return ret;
}

MyCSRMat initMatrix(MyCSRMat myMat, const char * myMatName)
{
    int ret_code;
    MM_typecode matcode;
    FILE *f;
    int nz;
    int M, N;
    int i, *K, *I, *J;
    double *val;


    char buff[FILENAME_MAX];
    GetCurrentDir( buff, FILENAME_MAX );
    const char string[] = "/examples/testdata/";
    strcat(buff,string);
    strcat(buff,myMatName);
    printf("matrix ");
    printf(myMatName);
    printf(" read at : \n");
    printf(buff);
    printf("\n");


    if ((f = fopen(buff, "r")) == NULL)
        exit(1);


    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        exit(1);
    }

    /*  This is how one can screen matrix types if their application */
    /*  only supports a subset of the Matrix Market data types.      */

    if (mm_is_complex(matcode) && mm_is_matrix(matcode) &&
            mm_is_sparse(matcode) )
    {
        printf("Sorry, this application does not support ");
        printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
        exit(1);
    }

    /* find out size of sparse matrix .... */
    if ((ret_code = mm_read_mtx_crd_size(f, &M, &N, &nz)) !=0)
        exit(1);


    /* reseve memory for matrices */

    K = (int *) malloc(nz * sizeof(int));
    I = (int *) malloc(nz+1 * sizeof(int)); // +1 because we put the number of nz in the end
    J = (int *) malloc(nz * sizeof(int));
    val = (double *) malloc(nz * sizeof(double));


    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */
    for (i=0; i<nz; i++)
    {
        fscanf(f, "%d %d %lg\n", &K[i], &J[i], &val[i]);
        K[i];  /* adjust from 1-based to 0-based */
        J[i];
    }

    I = K;
    I[nz] = M;

    //printf("I[nz] : %d\n",I[nz]);
    //printf("I[nz-1] : %d\n",I[nz-1]);
    //printf("J[nz-1] : %d\n",J[nz-1]);
    //printf("val[nz-1] : %20.19g\n",val[nz-1]);

    if (f !=stdin) fclose(f);

    /************************/
    /* now write out matrix */
    /************************/

    mm_write_banner(stdout, matcode);
    mm_write_mtx_crd_size(stdout, M, N, nz);
    //for (i=0; i<nz; i++)
    //    fprintf(stdout, "%d %d %20.19g\n", I[i], J[i], val[i]);

    // myMat

    myMat.I = I;
    myMat.J = J;
    myMat.M = M;
    myMat.N = N;
    myMat.nz = nz;
    myMat.val = val;

    return myMat;

}

int compute(MyCSRMat myMat1, MyCSRMat myMat2)
{
    bool test = false; // TRUE => exemple | False => loaded matrix

    int N = 50000;
    // matrix generation and validation depends on these relationships:
    int SCL = 2;
    int K = N;
    int M = SCL*N;
    // A: MxK  B: KxN  C: MxN

    std::clock_t start;
    double duration, computeT;

    hipsparseStatus_t stat;
    hipsparseHandle_t hndl;
    hipsparseMatDescr_t descrA, descrB, descrC;
    int *csrRowPtrA, *csrRowPtrB, *csrRowPtrC, *csrColIndA, *csrColIndB, *csrColIndC;
    int *h_csrRowPtrA, *h_csrRowPtrB, *h_csrRowPtrC, *h_csrColIndA, *h_csrColIndB, *h_csrColIndC;
    float *csrValA, *csrValB, *csrValC, *h_csrValA, *h_csrValB, *h_csrValC;
    double *h_csrValCd;
    int nnzA, nnzB, nnzC;   // number of non-zero
    int m,n,k;
    m = M;
    n = N;
    k = K;

    if (test){
        ///////////////////////////     STEP 1      ////////////////////////////////////
        /* -------------------------------------------------------------------------- */
        /*                           generate A, B=2I                                 */
        /*
                                     A:
                                    |1.0 0.0 0.0 ...|
                                    |1.0 0.0 0.0 ...|
                                    |0.0 1.0 0.0 ...|
                                    |0.0 1.0 0.0 ...|
                                    |0.0 0.0 1.0 ...|
                                    |0.0 0.0 1.0 ...|
                                    ...

                                    B:
                                    |2.0 0.0 0.0 ...|
                                    |0.0 2.0 0.0 ...|
                                    |0.0 0.0 2.0 ...|
                                    ...                                               */
        /* -------------------------------------------------------------------------- */
        start = std::clock();

            nnzA = m;
            nnzB = n;
            h_csrRowPtrA = (int *)malloc((nnzA+1)*sizeof(int));
            h_csrColIndA = (int *)malloc(nnzA*sizeof(int));
            h_csrValA  = (float *)malloc(nnzA*sizeof(float));

            h_csrRowPtrB = (int *)malloc((nnzB+1)*sizeof(int));
            h_csrColIndB = (int *)malloc(nnzB*sizeof(int));
            h_csrValB  = (float *)malloc(nnzB*sizeof(float));

        duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
        printf("Host Malloc :                                   %f ms\n", duration);
        //------------------------------------------------------------------------------
        start = std::clock();

            if ((h_csrRowPtrA == NULL) || (h_csrRowPtrB == NULL) || (h_csrColIndA == NULL) || (h_csrColIndB == NULL) || (h_csrValA == NULL) || (h_csrValB == NULL))
            {printf("malloc fail\n"); return -1;}
            for (int i = 0; i < nnzA; i++){
            h_csrValA[i] = 1.0f;
            h_csrRowPtrA[i] = i;
            h_csrColIndA[i] = i/SCL;
            if (i < nnzB){
              h_csrValB[i] = 2.0f;
              h_csrRowPtrB[i] = i;
              h_csrColIndB[i] = i;}
            }
            h_csrRowPtrA[nnzA] = nnzA;
            h_csrRowPtrB[nnzB] = nnzB;


        duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
        printf("CSR Matrix Generation :                         %f ms\n", duration);
    }
    else{
        nnzA = myMat1.nz;
        nnzB = myMat2.nz;
    }

    ///////////////////////////     STEP 2      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                      Allocate memory on the device                         */
    /*              and return a ptr of its memory emplacement                    */
    /* -------------------------------------------------------------------------- */

    if (test){
        start = std::clock();

            hipMalloc(&csrRowPtrA, (m+1)*sizeof(int));

        duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
        printf("First hipMalloc :                              %f ms\n", duration);
        //------------------------------------------------------------------------------
        start = std::clock();

            hipMalloc(&csrColIndA, nnzA*sizeof(int));
            hipMalloc(&csrValA, nnzA*sizeof(float));

            hipMalloc(&csrRowPtrB, (nnzB+1)*sizeof(int));
            hipMalloc(&csrColIndB, nnzB*sizeof(int));
            hipMalloc(&csrValB, nnzB*sizeof(float));

        duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
        printf("hipMalloc csrRowPtrB|csrColIndA/B|csrValA/B :  %f ms\n", duration);
    }
    else {
        start = std::clock();

            hipMalloc(&csrRowPtrA, (myMat1.nz+1)*sizeof(int));

        duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
        printf("hipMalloc csrRowPtrA :                         %f ms\n", duration);
        //------------------------------------------------------------------------------
        start = std::clock();

            hipMalloc(&csrColIndA, myMat1.nz*sizeof(int));
            hipMalloc(&csrValA, myMat1.nz*sizeof(double));

            hipMalloc(&csrRowPtrB, (myMat2.nz+1)*sizeof(int));
            hipMalloc(&csrColIndB, myMat2.nz*sizeof(int));
            hipMalloc(&csrValB, myMat2.nz*sizeof(double));


        duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
        printf("hipMalloc csrRowPtrB|csrColIndA/B|csrValA/B :  %f ms\n", duration);
    }

    ///////////////////////////     STEP 3      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                  Copy the data from the Host (CPU)                         */
    /*                      to the device (GPU)                                   */
    /* -------------------------------------------------------------------------- */
    start = std::clock();
    computeT = start;

    if (test){
        cudaCheckErrors("hipMalloc fail");
        hipMemcpy(csrRowPtrA, h_csrRowPtrA, (nnzA+1)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrColIndA, h_csrColIndA, nnzA*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrValA, h_csrValA, nnzA*sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(csrRowPtrB, h_csrRowPtrB, (nnzB+1)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrColIndB, h_csrColIndB, nnzB*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrValB, h_csrValB, nnzB*sizeof(float), hipMemcpyHostToDevice);
        cudaCheckErrors("hipMemcpy fail");
    }
    else{
        cudaCheckErrors("hipMalloc fail");
        hipMemcpy(csrRowPtrA, myMat1.I, (myMat1.nz+1)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrColIndA, myMat1.J, myMat1.nz*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrValA, myMat1.val, myMat1.nz*sizeof(double), hipMemcpyHostToDevice);

        hipMemcpy(csrRowPtrB, myMat2.I, (myMat2.nz+1)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrColIndB, myMat2.J, myMat2.nz*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(csrValB, myMat2.val, myMat2.nz*sizeof(double), hipMemcpyHostToDevice);
        cudaCheckErrors("hipMemcpy fail");

        //printf(myMat2.I);
    }

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("Copy Data from Host to Device :                 %f ms\n", duration);
    ///////////////////////////     STEP 4      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                      set cusparse matrix types                             */
    /*                             ?????                                          */
    /* -------------------------------------------------------------------------- */
    start = std::clock();

        CUSPARSE_CHECK(hipsparseCreate(&hndl));

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("hipsparseCreate(&hndl) :                         %f ms\n", duration);
    //------------------------------------------------------------------------------
    start = std::clock();

        stat = hipsparseCreateMatDescr(&descrA);
        CUSPARSE_CHECK(stat);
        stat = hipsparseCreateMatDescr(&descrB);
        CUSPARSE_CHECK(stat);
        stat = hipsparseCreateMatDescr(&descrC);
        CUSPARSE_CHECK(stat);

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("hipsparseCreateMatDescr(&descrA/B/C) :           %f ms\n", duration);
    //------------------------------------------------------------------------------
    start = std::clock();

        stat = hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
        CUSPARSE_CHECK(stat);
        stat = hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
        CUSPARSE_CHECK(stat);
        stat = hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
        CUSPARSE_CHECK(stat);
        stat = hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
        CUSPARSE_CHECK(stat);
        stat = hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
        CUSPARSE_CHECK(stat);
        stat = hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
        CUSPARSE_CHECK(stat);
        hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;


    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("Set cusparse matrix types :                     %f ms\n", duration);
    ///////////////////////////     STEP 5      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                          Determine csrRowPtrC                              */
    /*                                   &                                        */
    /*                  the total number of nonzero elements                      */
    /*                                                                            */
    /* -------------------------------------------------------------------------- */
    start = std::clock();

    // figure out size of C
    int baseC;

    if (test){
        // nnzTotalDevHostPtr points to host memory
        int *nnzTotalDevHostPtr = &nnzC;
        stat = hipsparseSetPointerMode(hndl, HIPSPARSE_POINTER_MODE_HOST);
        CUSPARSE_CHECK(stat);
        hipMalloc((void**)&csrRowPtrC, sizeof(int)*(m+1));
        cudaCheckErrors("hipMalloc fail");

    //------------------------------------------------------------------------------

        // ????
        stat = hipsparseXcsrgemmNnz(hndl, transA, transB, m, n, k,
            descrA, nnzA, csrRowPtrA, csrColIndA,
            descrB, nnzB, csrRowPtrB, csrColIndB,
            descrC, csrRowPtrC, nnzTotalDevHostPtr );
        CUSPARSE_CHECK(stat);

    //------------------------------------------------------------------------------

        //  ????
        if (NULL != nnzTotalDevHostPtr){
        nnzC = *nnzTotalDevHostPtr;}
        else{
        hipMemcpy(&nnzC, csrRowPtrC+m, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy fail");
        nnzC -= baseC;}
        hipMalloc((void**)&csrColIndC, sizeof(int)*nnzC);
        hipMalloc((void**)&csrValC, sizeof(float)*nnzC);
        cudaCheckErrors("hipMalloc fail");
    }
    else{
        // nnzTotalDevHostPtr points to host memory
        int *nnzTotalDevHostPtr = &nnzC;
        printf("INFO 1 NNZC:                                    %i\n",nnzC);
        stat = hipsparseSetPointerMode(hndl, HIPSPARSE_POINTER_MODE_HOST);
        CUSPARSE_CHECK(stat);
        hipMalloc((void**)&csrRowPtrC, sizeof(int)*(myMat1.M+1));
        cudaCheckErrors("hipMalloc fail");

    //------------------------------------------------------------------------------

        // Determine csrRowPtrC
        stat = hipsparseXcsrgemmNnz(hndl, transA, transB, myMat1.M, myMat2.N, myMat1.N,
            descrA, nnzA, csrRowPtrA, csrColIndA,
            descrB, nnzB, csrRowPtrB, csrColIndB,
            descrC, csrRowPtrC, nnzTotalDevHostPtr );
        CUSPARSE_CHECK(stat);

    //------------------------------------------------------------------------------

        //  Gathers nnzC
        if (NULL != nnzTotalDevHostPtr){
        nnzC = *nnzTotalDevHostPtr;}
        else
        {
            hipMemcpy(&nnzC, csrRowPtrC+myMat1.M, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&baseC, csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
            cudaCheckErrors("hipMemcpy fail");
            nnzC -= baseC;
        }
        hipMalloc((void**)&csrColIndC, sizeof(int)*nnzC);
        hipMalloc((void**)&csrValC, sizeof(double)*nnzC);
        cudaCheckErrors("hipMalloc fail");
        printf("INFO 2 NNZC:                                    %i\n",nnzC);

    }

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("nnzTotalDevHostPtr points to host memory :      %f ms\n", duration);
    ///////////////////////////     STEP 6      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                Perform multiplication C = A*B                              */
    /*                                                                            */
    /* -------------------------------------------------------------------------- */
    start = std::clock();

    if (test){
        stat = hipsparseScsrgemm(hndl, transA, transB, m, n, k,
            descrA, nnzA,
            csrValA, csrRowPtrA, csrColIndA,
            descrB, nnzB,
            csrValB, csrRowPtrB, csrColIndB,
            descrC,
            csrValC, csrRowPtrC, csrColIndC);
        CUSPARSE_CHECK(stat);
    }
    else{
        stat = hipsparseScsrgemm(hndl, transA, transB, myMat1.M, myMat2.N, myMat1.N,
            descrA, nnzA,
            csrValA, csrRowPtrA, csrColIndA,
            descrB, nnzB,
            csrValB, csrRowPtrB, csrColIndB,
            descrC,
            csrValC, csrRowPtrC, csrColIndC);
        CUSPARSE_CHECK(stat);
    }

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("GPU calculation time :                          %f ms\n", duration);
    ///////////////////////////     STEP 7      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                  Copy result (C) back to host                              */
    /*                       & test & validate it                                 */
    /* -------------------------------------------------------------------------- */
    start = std::clock();
    if (test){
        // copy result (C) back to host
        h_csrRowPtrC = (int *)malloc((m+1)*sizeof(int));
        h_csrColIndC = (int *)malloc(nnzC *sizeof(int));
        h_csrValC  = (float *)malloc(nnzC *sizeof(float));
        if ((h_csrRowPtrC == NULL) || (h_csrColIndC == NULL) || (h_csrValC == NULL))
        {printf("malloc fail\n"); return -1;}
        hipMemcpy(h_csrRowPtrC, csrRowPtrC, (m+1)*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_csrColIndC, csrColIndC,  nnzC*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_csrValC, csrValC, nnzC*sizeof(float), hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy fail");

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("Copy GPU to CPU :                               %f ms\n", duration);
    //------------------------------------------------------------------------------


        // check result, C = 2A
        if (nnzC != m) {printf("invalid matrix size C: %d, should be: %d\n", nnzC, m); return -1;}
        for (int i = 0; i < m; i++){
        if (h_csrRowPtrA[i] != h_csrRowPtrC[i]) {printf("A/C row ptr mismatch at %d, A: %d, C: %d\n", i, h_csrRowPtrA[i], h_csrRowPtrC[i]); return -1;}
        if (h_csrColIndA[i] != h_csrColIndC[i]) {printf("A/C col ind mismatch at %d, A: %d, C: %d\n", i, h_csrColIndA[i], h_csrColIndC[i]); return -1;}
        if ((h_csrValA[i]*2.0f) != h_csrValC[i]) {printf("A/C value mismatch at %d, A: %f, C: %f\n", i, h_csrValA[i]*2.0f, h_csrValC[i]); return -1;}
        }
    }
    else{
        // copy result (C) back to host
        h_csrRowPtrC = (int *)malloc((myMat1.M+1)*sizeof(int));
        h_csrColIndC = (int *)malloc(nnzC *sizeof(int));
        h_csrValCd  = (double *)malloc(nnzC *sizeof(double));
        if ((h_csrRowPtrC == NULL) || (h_csrColIndC == NULL) || (h_csrValCd == NULL))
        {printf("malloc fail\n"); return -1;}
        hipMemcpy(h_csrRowPtrC, csrRowPtrC, (myMat1.M+1)*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_csrColIndC, csrColIndC, nnzC*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_csrValCd, csrValC, nnzC*sizeof(double), hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy fail");

    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("Copy GPU to CPU :                               %f ms\n", duration);
    //------------------------------------------------------------------------------

        //if (nnzC != myMat1.M) {printf("invalid matrix size C: %d, should be: %d\n", nnzC, myMat1.M); return -1;}

    }
    duration = (( std::clock() - computeT ) / (double) CLOCKS_PER_SEC) * 1000;
    printf("===========================================================\n");
    printf("RESULT PRODUCT INFO NZ:                         %i\n",nnzC);
    printf("REAL TIME TO COMPUTE :                          %f ms\n", duration);

    myMat3.I = h_csrRowPtrC;
    myMat3.J = h_csrColIndC;
    myMat3.M = myMat1.M+1;
    myMat3.N = nnzC;
    myMat3.nz = nnzC;
    myMat3.val = h_csrValCd;

//    for (int i=0; i<myMat3.nz; i++)
//        fprintf(stdout, "%d %d %d\n", myMat3.I[i], myMat3.J[i], myMat3.val[i]);


    return 0;
}

inline double exec_time_ms(double time_sc){
    return time_sc*1000;
}

// perform sparse-matrix multiplication C=AxB
int main(int argc, char *argv[]){

//    int ret_code;
//    MM_typecode matcode;
//    FILE *f;
//    int nz;
//    int M, N;
//    int i, *K, *I, *J;
//    double *val;


//    if (argc < 3)
//    {
//        fprintf(stderr, "Usage: %s [martix-market-filename]\n", argv[0]);
//        exit(1);
//    }

//    myMat1 = initMatrix(myMat1,argv[1]);
//    myMat2 = initMatrix(myMat2,argv[2]);

//    //printf("TEST :\n");
//    //printf("I[nz] : %d\n",myMat1.I[myMat1.nz]);
//    //printf("I[nz-1] : %d\n",myMat1.I[myMat1.nz-1]);
//    //printf("J[nz-1] : %d\n",myMat1.J[myMat1.nz-1]);
//    //printf("val[nz-1] : %20.19g\n",myMat1.val[myMat1.nz-1]);

//    //printf("I[nz] : %d\n",myMat2.I[myMat2.nz]);
//    //printf("I[nz-1] : %d\n",myMat2.I[myMat2.nz-1]);
//    //printf("J[nz-1] : %d\n",myMat2.J[myMat2.nz-1]);
//    //printf("val[nz-1] : %20.19g\n",myMat2.val[myMat2.nz-1]);

//    std::clock_t start;
//    double duration;
//    printf("----------          COMPUTE K * K       ---------------------\n\n");
//    start = std::clock();
//    compute(myMat1,myMat2);
//    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
//    printf("COMPLETE PROCESS 1 TIME :                       %f ms\n", duration);

//    printf("\n-----------------------------------------------------------\n\n");

//    start = std::clock();
//    compute(myMat1,myMat2);
//    duration = (( std::clock() - start ) / (double) CLOCKS_PER_SEC) * 1000;
//    printf("COMPLETE PROCESS 2 TIME :                       %f ms\n", duration);


//    return 0;


///////////////////////////////////////////
///////////////////////////////////////////



    char buff[FILENAME_MAX];
    GetCurrentDir( buff, FILENAME_MAX );
    std::string testdata_dir(buff);
    testdata_dir.append("/examples/testdata");
    std::string myMatrix1 = testdata_dir;
    std::string myMatrix2 = testdata_dir;
    myMatrix1.append("/matKeig.mtx");
    myMatrix2.append("/matJ1eig.mtx");
    bool info = true;
    int benchmarkNbrRun = 100;
    typedef double ScalarType;

    viennacl::tools::timer timer;
    double exec_time_read,exec_time_read_ublas,exec_time_copy,exec_time;

    int sizeK = 46659; //20000;//
    int sizeJ = 31; //30;//


    ///////////////////////////     STEP 1      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                       Init all the variables                               */
    /* -------------------------------------------------------------------------- */

    ////////////////////////////////////////////////////////////////////////////////
    ///  VIENNACL SPARSE MATRIX

    viennacl::compressed_matrix<ScalarType> vcl_compressed_JtKJ;
    viennacl::compressed_matrix<ScalarType> vcl_compressed_K;
    viennacl::compressed_matrix<ScalarType> vcl_compressed_KJ(sizeK,sizeJ);
    viennacl::compressed_matrix<ScalarType> vcl_compressed_J;
    viennacl::compressed_matrix<ScalarType> vcl_compressed_Jt;

    ////////////////////////////////////////////////////////////////////////////////
    ///  UBLAS SPARSE MATRIX

    boost::numeric::ublas::compressed_matrix<ScalarType> ublas_K(sizeK,sizeK);
    boost::numeric::ublas::compressed_matrix<ScalarType> ublas_J(sizeK,sizeJ);
    boost::numeric::ublas::compressed_matrix<ScalarType> ublas_Jt(sizeJ,sizeK);


    ///////////////////////////     STEP 2      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                  Read from file the sparse matrix                          */
    /*                   (in the Matrix Market format)                            */
    /* -------------------------------------------------------------------------- */

    ////////////////////////////////////////////////////////////////////////////////
    ///  WITH UBLAS

    timer.start();
    if (!viennacl::io::read_matrix_market_file(ublas_K, myMatrix1))
    {
      std::cout << "Error reading Matrix file" << std::endl;
      return 0;
    }
    //unsigned int cg_mat_size = cg_mat.size();
    std::cout << "done reading K" << std::endl;

    if (!viennacl::io::read_matrix_market_file(ublas_J, myMatrix2))
    {
      std::cout << "Error reading Matrix file" << std::endl;
      return 0;
    }
    //unsigned int cg_mat_size = cg_mat.size();
    std::cout << "done reading J" << std::endl;
    exec_time_read_ublas = timer.get();


    ///////////////////////////     STEP 3      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                  Copy matrix from CPU to GPU                               */
    /* -------------------------------------------------------------------------- */
    timer.start();

    // With UBlas Matrix

    ublas_Jt = boost::numeric::ublas::trans(ublas_J);
    viennacl::copy(ublas_J, vcl_compressed_J);
    viennacl::copy(ublas_Jt, vcl_compressed_Jt);
    timer.start();

    viennacl::copy(ublas_K, vcl_compressed_K);

    exec_time_copy = timer.get();

    if (info)
    {
        std::cout << "\n" <<"      ublas_K (rows,cols) : "<< ublas_K.size1() << " " << ublas_K.size2() << std::endl;
        std::cout << "      ublas_J (rows,cols) : "<< ublas_J.size1() << " " << ublas_J.size2() << std::endl;
        std::cout << "      ublas_Jt (rows,cols) : "<< ublas_Jt.size1() << " " << ublas_Jt.size2() << std::endl;
        std::cout << "      ----------------------------------------------\n" << std::endl;
        std::cout << "      Time to copy CPU->GPU : " << exec_time_ms(exec_time_copy) << " ms\n" << std::endl;
    }

    ///////////////////////////     STEP 4      ////////////////////////////////////
    /* -------------------------------------------------------------------------- */
    /*                         Matrix operations                                  */
    /*                             Jt * K J                                       */
    /* -------------------------------------------------------------------------- */


    //  On GPU with ViennaCL Compressed Matrix
    std::cout << "      ------- Jt*K*J product on GPU ----------" << std::endl;
    viennacl::backend::finish();
    timer.start();
    for (int runs=0; runs<benchmarkNbrRun; ++runs)
    {
        vcl_compressed_KJ = viennacl::linalg::prod(vcl_compressed_K,vcl_compressed_J);
        vcl_compressed_JtKJ = viennacl::linalg::prod(vcl_compressed_Jt,vcl_compressed_KJ);
    }
    viennacl::backend::finish();
    exec_time = timer.get();
    std::cout << "      GPU time align1: " << exec_time_ms(exec_time) << " ms\n"<< std::endl;

    return 0;

}
